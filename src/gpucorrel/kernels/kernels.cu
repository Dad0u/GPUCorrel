#include "hip/hip_runtime.h"
/*
This file defines kernels and textures used in TechCorrel
to perform GPU-accelerated correlation with pyCUDA.
All the kernels must be call with a block*grid size superior or
equal to the dimension of the output
ex: for a 2048*2048 image, block=(32,32,1) grid=(64,64)
*/

#include <hip/hip_runtime.h>
#define WIDTH %d
#define HEIGHT %d
#define PARAMETERS %d // The number of fields

// Original image
texture<float, hipTextureType2D, hipReadModeElementType> tex;
// Second image
texture<float, hipTextureType2D, hipReadModeElementType> tex_d;
// The mask, to limit the effect of the borders
texture<float, hipTextureType2D, hipReadModeElementType> texMask;

// This kernel computes the gradients of the reference image with a Sobel filter
// Note: the outputs are not normalized
__global__ void gradient(float* gradX, float* gradY)
{
  const unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
  if(x < WIDTH && y < HEIGHT)
  {
    gradX[x+WIDTH*y] = (
        tex2D(tex, (x+1.5f)/WIDTH, (float)y/HEIGHT)
        +tex2D(tex, (x+1.5f)/WIDTH, (y+1.f)/HEIGHT)
        -tex2D(tex, (x-.5f)/WIDTH, (float)y/HEIGHT)
        -tex2D(tex, (x-.5f)/WIDTH, (y+1.f)/HEIGHT)
         );
    gradY[x+WIDTH*y] = (
        tex2D(tex, (float)x/WIDTH, (y+1.5f)/HEIGHT)
        +tex2D(tex, (x+1.f)/WIDTH, (y+1.5f)/HEIGHT)
        -tex2D(tex, (float)x/WIDTH, (y-.5f)/HEIGHT)
        -tex2D(tex, (x+1.f)/WIDTH, (y-.5f)/HEIGHT)
        );
  }
}

// Kernel to resample the reference image using bilinear interpolation
__global__ void resampleR(float* out, const int w, const int h)
{
  const int idx = threadIdx.x+blockIdx.x*blockDim.x;
  const int idy = threadIdx.y+blockIdx.y*blockDim.y;
  if(idx < w && idy < h)
    out[idx+w*idy] = tex2D(tex,(float)idx/w,(float)idy/h);
}

// To resample the second image...
__global__ void resample(float* out, const int w, const int h)
{
  const int idx = threadIdx.x+blockIdx.x*blockDim.x;
  const int idy = threadIdx.y+blockIdx.y*blockDim.y;
  if(idx < w && idy < h)
    out[idx+w*idy] = tex2D(tex_d,(float)idx/w,(float)idy/h);
}

// This kernel computes the tables that will be used by the correlation
// routine to evaluate the research direction (called G, 1 per parameter)
__global__ void makeG(float* G, const float* gradX, const float* gradY,
                      const float* fieldX, const float* fieldY)
{
  const int idx = threadIdx.x+blockIdx.x*blockDim.x;
  const int idy = threadIdx.y+blockIdx.y*blockDim.y;
  if(idx < WIDTH && idy < HEIGHT)
  {
    int id = idx+WIDTH*idy;
    G[id] = gradX[id]*fieldX[id]+gradY[id]*fieldY[id];
  }
}

// The kernel that will write the residual image (the difference between the
// reference image and the second image after deformation)
__global__ void makeDiff(float *out, const float *param,
                         const float *fieldsX, const float *fieldsY)
{
  const int idx = threadIdx.x+blockIdx.x*blockDim.x;
  const int idy = threadIdx.y+blockIdx.y*blockDim.y;
  if(idx < WIDTH && idy < HEIGHT)
  {
    const int id = idx+WIDTH*idy;
    float ox = .5f;
    float oy = .5f;
    // First, let's compute the offset we have by adding all the fields
    for(unsigned int i = 0; i < PARAMETERS; i++)
    {
      ox += param[i]*fieldsX[WIDTH*HEIGHT*i+id];
      oy += param[i]*fieldsY[WIDTH*HEIGHT*i+id];
    }
    // The residual in idx,idy is the value of the reference image
    // minus the value of the second image at the new coordinates
    // We multiply this difference by the mask
    out[id] = (
    tex2D(tex,(idx+.5f)/WIDTH,(idy+.5f)/HEIGHT)
    -tex2D(tex_d,(idx+ox)/WIDTH,(idy+oy)/HEIGHT)
    )*tex2D(texMask,idx+.5f,idy+.5f);
  }
}

// Simple matrix-vector dot product (to multiply the inverted Hessian with
// the research direction)
__global__ void myDot(const float *M, float *v)
{
  unsigned int id = threadIdx.x;
  __shared__ float sh_v[PARAMETERS];
  float val = 0;
  sh_v[id] = v[id];
  __syncthreads();
  for(unsigned int i = 0; i < PARAMETERS; i++)
  {
    val += M[id*PARAMETERS+i]*sh_v[i];
  }
  v[id] = val;
}

// Do I really need to explain this one ?
__global__ void kadd(float* v, const float k, const float* v2)
{
  v[threadIdx.x] += k*v2[threadIdx.x];
}
